#include "hip/hip_runtime.h"
/**
 *
 *  Copyright 2016-2020 Netflix, Inc.
 *  Copyright 2021 NVIDIA Corporation.
 *
 *     Licensed under the BSD+Patent License (the "License");
 *     you may not use this file except in compliance with the License.
 *     You may obtain a copy of the License at
 *
 *         https://opensource.org/licenses/BSDplusPatent
 *
 *     Unless required by applicable law or agreed to in writing, software
 *     distributed under the License is distributed on an "AS IS" BASIS,
 *     WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *     See the License for the specific language governing permissions and
 *     limitations under the License.
 *
 */

#include "cuda_helper.cuh"
#include "integer_adm_kernels.h"

#include <assert.h>

template <int cols_per_thread>
static __device__ __forceinline__ void copy_vec_4(const int32_t * __restrict__ in, int32_t * __restrict__ out)
{
  __builtin_assume_aligned(in, 16);
  __builtin_assume_aligned(out, 16);

  static_assert(cols_per_thread % 4 == 0, "implemented only for a multiple of 4");
  #pragma unroll
  for (int col = 0;col <cols_per_thread;col += 4) {
    *reinterpret_cast<uint4*>(out + col) = *reinterpret_cast<const uint4*>(in + col);
  }
}

template <int cols_per_thread>
static __device__ __forceinline__ void copy_vec_4(const int16_t * __restrict__ in, int16_t * __restrict__ out)
{
  __builtin_assume_aligned(in, 8);
  __builtin_assume_aligned(out, 8);

  static_assert(cols_per_thread % 4 == 0, "implemented only for a multiple of 4");
  #pragma unroll
  for (int col = 0;col <cols_per_thread;col += 4) {
    *reinterpret_cast<ushort4*>(out + col) = *reinterpret_cast<const ushort4*>(in + col);
  }
}

template <int rows_per_thread, int cols_per_thread>
__global__ void i4_adm_csf_kernel(AdmBufferCuda buf, int scale, int top,
                                  int bottom, int left, int right, int stride,
                                  AdmFixedParametersCuda params) {

  int band = blockIdx.z + 1;
  const int32_t *src_ptr = buf.i4_decouple_a.bands[band]; // this is evaluated to a LDC
  int32_t *dst_ptr = buf.i4_csf_a.bands[band];
  int32_t *flt_ptr = buf.i4_csf_f.bands[band];

  int y = top + (blockIdx.y * blockDim.y + threadIdx.y) * rows_per_thread;
  int x = left + (blockIdx.x * blockDim.x + threadIdx.x) * cols_per_thread;

  const uint32_t i_rfactor = params.i_rfactor[scale * 3 + blockIdx.z];
  const uint32_t FIX_ONE_BY_30 = 143165577;
  const uint32_t shift_dst = 28;
  const uint32_t shift_flt = 32;
  const int32_t add_bef_shift_dst = (1u << (shift_dst - 1));
  const int32_t add_bef_shift_flt = (1u << (shift_flt - 1));

  const int offset = y * stride + x;

  if (y < bottom && x < right) {
    __align__(16) int32_t src[cols_per_thread];
    __align__(16) int32_t dst_vec[cols_per_thread];
    __align__(16) int32_t flt_vec[cols_per_thread];

    for (int row = 0;row < rows_per_thread;++row) {
      copy_vec_4<cols_per_thread>(src_ptr + offset + row * stride, src);

      for (int col = 0;col < cols_per_thread;++col) {
          int32_t dst_val = (int32_t)(((i_rfactor * int64_t(src[col])) +
                                      add_bef_shift_dst) >>
                                      shift_dst);
          dst_vec[col] = dst_val;
          flt_vec[col] = (int32_t)((((int64_t)FIX_ONE_BY_30 * abs(dst_val)) +
                                          add_bef_shift_flt) >>
                                          shift_flt);
      }
      copy_vec_4<cols_per_thread>(dst_vec, dst_ptr + offset + row * stride);
      copy_vec_4<cols_per_thread>(flt_vec, flt_ptr + offset + row * stride);
    }
  }
}

__constant__ const uint8_t i_shifts[4] = {0, 15, 15, 17};
__constant__ const uint16_t i_shiftsadd[4] = {0, 16384, 16384, 65535};

template <int rows_per_thread, int cols_per_thread>
__global__ void adm_csf_kernel(AdmBufferCuda buf, int top, int bottom, int left,
                               int right, int stride,
                               AdmFixedParametersCuda params) {
  const int band = blockIdx.z + 1;

  const int16_t *src_ptr = buf.decouple_a.bands[band]; // this is evaluated to a LDC
  int16_t *dst_ptr = buf.csf_a.bands[band];
  int16_t *flt_ptr = buf.csf_f.bands[band];
  int y = top + (blockIdx.y * blockDim.y + threadIdx.y) * rows_per_thread;
  int x = left + (blockIdx.x * blockDim.x + threadIdx.x) * cols_per_thread;

  const uint32_t i_rfactor = params.i_rfactor[blockIdx.z];
  const uint16_t FIX_ONE_BY_30 = 4369; //(1/30)*2^17

  if (y < bottom && x < right) {
    __align__(8) int16_t src[cols_per_thread];
    __align__(8) int16_t dst_vec[cols_per_thread];
    __align__(8) int16_t flt_vec[cols_per_thread];

    const int offset = y * stride + x;

    for (int row = 0;row < rows_per_thread;++row) {
      copy_vec_4<cols_per_thread>(src_ptr + offset + row * stride, src);

      for (int col = 0;col < cols_per_thread;++col) {
        int32_t dst_val = i_rfactor * (uint32_t)src[col];
        int16_t i16_dst_val = (dst_val + i_shiftsadd[band]) >> i_shifts[band];
        dst_vec[col] = i16_dst_val;
        flt_vec[col] =
            ((FIX_ONE_BY_30 * abs((int32_t)i16_dst_val)) + 2048) >> 12;
      }
      copy_vec_4<cols_per_thread>(dst_vec, dst_ptr + offset + row * stride);
      copy_vec_4<cols_per_thread>(flt_vec, flt_ptr + offset + row * stride);
    }
  }
}

extern "C" {
void i4_adm_csf_device(AdmBufferCuda *buf, int scale, int w, int h, int stride,
                       AdmFixedParametersCuda *p, hipStream_t c_stream) {
  // ensure that all pointers are aligned to 16 bytes for vectorized memory access
  for (int band = 0;band < 3;++band) {
    assert((reinterpret_cast<size_t>(buf->i4_decouple_a.bands[band]) & 15) == 0); 
    assert((reinterpret_cast<size_t>(buf->i4_csf_a.bands[band]) & 15) == 0);
    assert((reinterpret_cast<size_t>(buf->i4_csf_f.bands[band]) & 15) == 0); 
  }

  // ensure that the stride is a multiple of 4 so that each row starts 16 byte aligned.
  assert(stride % 4 == 0);

  /* The computation of the score is not required for the regions
  which lie outside the frame borders */
  int left = w * float(ADM_BORDER_FACTOR) - 0.5f - 1; // -1 for filter tap
  int top = h * float(ADM_BORDER_FACTOR) - 0.5f - 1;
  int right = w - left + 2; // +2 for filter tap
  int bottom = h - top + 2;

  if (left < 0) {
    left = 0;
  }
  if (right > w) {
    right = w;
  }
  if (top < 0) {
    top = 0;
  }
  if (bottom > h) {
    bottom = h;
  }

  // align left side to ensure that all memory accesses start at a multiple of 16 bytes.
  // this will do a little bit more work than originally requested, though the result is unchanged.
  left = left & ~3;

  const int cols_per_thread = 4;
  const int rows_per_thread = 1;
  dim3 block(32, 4);
  dim3 grid(DIV_ROUND_UP(right - left, block.x * cols_per_thread),
            DIV_ROUND_UP(bottom - top, block.y * rows_per_thread),
            3);
  i4_adm_csf_kernel<rows_per_thread, cols_per_thread><<<grid, block, 0, c_stream>>>(*buf, scale, top, bottom,
                                                  left, right, stride, *p);
  CudaCheckError();
}

void adm_csf_device(AdmBufferCuda *buf, int w, int h, int stride,
                    AdmFixedParametersCuda *p, hipStream_t c_stream) {
  // ensure that all pointers are aligned to 16 bytes for vectorized memory access
  for (int band = 0;band < 3;++band) {
    assert((reinterpret_cast<size_t>(buf->i4_decouple_a.bands[band]) & 15) == 0); 
    assert((reinterpret_cast<size_t>(buf->i4_csf_a.bands[band]) & 15) == 0);
    assert((reinterpret_cast<size_t>(buf->i4_csf_f.bands[band]) & 15) == 0); 
  }

  // ensure that the stride is a multiple of 4 so that each row starts 16 byte aligned.
  assert(stride % 4 == 0);

  /* The computation of the score is not required for the regions
  which lie outside the frame borders */
  int left = w * float(ADM_BORDER_FACTOR) - 0.5f - 1; // -1 for filter tap
  int top = h * float(ADM_BORDER_FACTOR) - 0.5f - 1;
  int right = w - left + 2; // +2 for filter tap
  int bottom = h - top + 2;

  if (left < 0) {
    left = 0;
  }
  if (right > w) {
    right = w;
  }
  if (top < 0) {
    top = 0;
  }
  if (bottom > h) {
    bottom = h;
  }

  // align left side to ensure that all memory accesses start at a multiple of 16 bytes.
  // this will do a little bit more work than originally requested, though the result is unchanged.
  left = left & ~3;

  const int cols_per_thread = 4;
  const int rows_per_thread = 1;
  dim3 block(32, 4);
  dim3 grid(DIV_ROUND_UP(right - left, block.x * cols_per_thread),
            DIV_ROUND_UP(bottom - top, block.y * rows_per_thread),
            3);

  adm_csf_kernel<rows_per_thread, cols_per_thread><<<grid, block, 0, c_stream>>>(*buf, top, bottom, left, right,
                                               stride, *p);
  CudaCheckError();
}
} // extern "C"
